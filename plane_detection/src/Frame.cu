
#include <hip/hip_runtime.h>
ushort *d_in;
float *d_out;

__global__ void _cudaReadPointCloud(
        ushort *d_in, 
        float *d_out, 
        const int width, 
        const int height
) {
    unsigned u = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned v = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned idx = width * v + u;

    float depth = (float)d_in[idx] / 5000.0f;
    d_out[idx * 4] = (u - 318.6f) * depth / 517.3f;  
    d_out[idx * 4 + 1] = (v - 255.3f) * depth / 516.5f;
    d_out[idx * 4 + 2] = depth;
    d_out[idx * 4 + 3] = 1.0f;
} 

extern void cudaReadPointCloud(
        ushort *h_in, 
        float *h_out,
        const int width, 
        const int height
) {
   
    dim3 blockSize = dim3(width / 40, height / 40);
    dim3 gridSize = dim3(40, 40);

    hipSetDevice(0);

    hipMalloc(&d_in, width * height * sizeof(ushort));
    hipMalloc(&d_out, 4 * width * height * sizeof(float));

    hipMemcpy(d_in, h_in, width * height * sizeof(ushort), hipMemcpyHostToDevice);
    _cudaReadPointCloud <<<gridSize, blockSize>>> (d_in, d_out, width, height);
    hipMemcpy(h_out, d_out, 4 * width * height * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}
